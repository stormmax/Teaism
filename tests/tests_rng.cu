

#include <hip/hip_runtime.h>
#include <iostream>
//#include <Cuda.h>
#include<hiprand/hiprand.h>
#include<hiprand/hiprand_kernel.h>


int n = 200;
using namespace std;

__device__ float generate( hiprandState* globalState, int ind ) 
{
    //int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}

__global__ void kernel(float* N, hiprandState* globalState, int n)
{
    // generate random numbers
    for(int i=0;i<40000;i++)
    {
        int k = generate(globalState, i) * 100000;
        while(k > n*n-1)
        {
            k-=(n*n-1);
        }
        N[i] = k;
    }
}

int main() 
{
    int N=40000;

    hiprandState* devStates;
    hipMalloc ( &devStates, N*sizeof( hiprandState ) );

    // setup seeds
    setup_kernel <<< 1, N >>> ( devStates,unsigned(time(NULL)) );

    float N2[40000];
    float* N3;
    hipMalloc((void**) &N3, sizeof(float)*N);

    kernel<<<1,1>>> (N3, devStates, n);

    hipMemcpy(N2, N3, sizeof(float)*N, hipMemcpyDeviceToHost);

    for(int i=0;i<N;i++)
    {
        cout<<N2[i]<<endl;
    }

    return 0;
}

